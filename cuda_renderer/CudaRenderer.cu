#include "hip/hip_runtime.h"
// ReSharper disable once CppPrecompiledHeaderIsNotIncluded
#include "CudaRenderer.cuh"
#include "CudaKernels.cuh"

#include "../common/Utils.hpp"

CudaRenderer::CudaRenderer(const RenderInfo* render_info, const WorldInfo* world_info, const SkyInfo* sky_info, const CameraInfo* camera_info)
	: render_info_(render_info), world_info_(world_info), sky_info_(sky_info), camera_info_(camera_info)
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
	blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
	threads_ = dim3(thread_x, thread_y);

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	CCE(hipMalloc(reinterpret_cast<void**>(&accumulation_buffer_), sizeof(float4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_initial_), sizeof(uint4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_state_), sizeof(uint4) * width * height));

	random_init<<<blocks_, threads_>>>(width, height, xoshiro_initial_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	CCE(hipMemcpy(xoshiro_state_, xoshiro_initial_, sizeof(uint4) * width * height, hipMemcpyDeviceToDevice));

	allocate_world();

	if (sky_info_->h_hdr_data)
	{
		const uint64_t hdr_size = sizeof(float3) * sky_info_->hdr_width * sky_info_->hdr_height;
		CCE(hipMalloc(reinterpret_cast<void**>(&sky_info_->d_hdr_data), hdr_size));
		CCE(hipMemcpy(sky_info_->d_hdr_data, sky_info_->h_hdr_data, hdr_size, hipMemcpyHostToDevice));
	}
}

CudaRenderer::~CudaRenderer()
{
	if (sky_info_->h_hdr_data)
		CCE(hipFree(sky_info_->d_hdr_data));

	deallocate_world();

	CCE(hipFree(xoshiro_state_));
	CCE(hipFree(xoshiro_initial_));
	CCE(hipFree(accumulation_buffer_));
	hipDeviceReset();
}

void CudaRenderer::render()
{
	const auto frame_buffer = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	if (render_info_->progressive)
	{
		render_pixel_progressive<<<blocks_, threads_>>>(frame_buffer, accumulation_buffer_, world_, *sky_info_, *render_info_, *camera_info_, xoshiro_state_);
	}
	else
	{
		CCE(hipMemset(frame_buffer, 0, render_info_->frame_size));

		for (int32_t i = 0; i < render_info_->samples_per_pixel; i++)
			render_pixel_static<<<blocks_, threads_>>>(frame_buffer, world_, *sky_info_, *render_info_, *camera_info_, xoshiro_state_);
	}

	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	CCE(hipFree(frame_buffer));
}

void CudaRenderer::refresh_buffer()
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;

	CCE(hipMemset(accumulation_buffer_, 0, sizeof(float4) * width * height));
	CCE(hipMemcpy(xoshiro_state_, xoshiro_initial_, sizeof(uint4) * width * height, hipMemcpyDeviceToDevice));
}

void CudaRenderer::refresh_texture(const int32_t index) const
{
	CCE(hipMemcpy(d_texture_data_ + index, &world_info_->textures_[index], sizeof(Texture), hipMemcpyHostToDevice));
}

void CudaRenderer::refresh_material(const int32_t index) const
{
	CCE(hipMemcpy(d_material_data_ + index, &world_info_->materials_[index], sizeof(Material), hipMemcpyHostToDevice));
}

void CudaRenderer::refresh_object(const int32_t index) const
{
	CCE(hipMemcpy(d_object_data_ + index, &world_info_->objects_[index], sizeof(Object), hipMemcpyHostToDevice));
}

void CudaRenderer::recreate_image()
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
	blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
	threads_ = dim3(thread_x, thread_y);

	CCE(hipFree(xoshiro_state_));
	CCE(hipFree(xoshiro_initial_));
	CCE(hipFree(accumulation_buffer_));
	CCE(hipMalloc(reinterpret_cast<void**>(&accumulation_buffer_), sizeof(float4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_initial_), sizeof(uint4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_state_), sizeof(uint4) * width * height));

	random_init<<<blocks_, threads_>>>(width, height, xoshiro_initial_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_sky()
{
	CCE(hipFree(sky_info_->d_hdr_data));

	if (sky_info_->h_hdr_data)
	{
		const uint64_t hdr_size = sizeof(float3) * sky_info_->hdr_width * sky_info_->hdr_height;
		CCE(hipMalloc(reinterpret_cast<void**>(&sky_info_->d_hdr_data), hdr_size));
		CCE(hipMemcpy(sky_info_->d_hdr_data, sky_info_->h_hdr_data, hdr_size, hipMemcpyHostToDevice));
	}
	else
		sky_info_->d_hdr_data = nullptr;
}

void CudaRenderer::map_frame_memory()
{
	const auto frame_buffer = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	CCE(hipMemcpy(render_info_->frame_data, frame_buffer, render_info_->frame_size, hipMemcpyDeviceToHost));
	CCE(hipFree(frame_buffer));
}

void CudaRenderer::allocate_world()
{
	auto& textures = world_info_->textures_;
	auto& materials = world_info_->materials_;
	auto& objects = world_info_->objects_;
	const auto texture_count = textures.size();
	const auto material_count = materials.size();
	const auto object_count = objects.size();

	const auto& texture_data = const_cast<Texture*>(textures.data());
	const auto& material_data = const_cast<Material*>(materials.data());
	const auto& object_data = const_cast<Object*>(objects.data());

	for (uint64_t i = 0; i < texture_count; i++)
	{
		if (textures[i].type == TextureType::IMAGE)
		{
			const auto image_data = &texture_data[i].image;
			const uint64_t image_size = sizeof(float) * image_data->width * image_data->height * 3;

			CCE(hipMalloc(reinterpret_cast<void**>(&image_data->d_data), image_size));
			CCE(hipMemcpy(image_data->d_data, image_data->h_data, image_size, hipMemcpyHostToDevice));
		}
	}

	for (uint64_t i = 0; i < object_count; i++)
	{
		if (objects[i].type == ObjectType::MODEL)
		{
			const auto model_data = &object_data[i].model;

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_vertices), model_data->vertex_count * sizeof(float3)));
			CCE(hipMemcpy(model_data->d_vertices, model_data->h_vertices, model_data->vertex_count * sizeof(float3), hipMemcpyHostToDevice));

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_indices), model_data->index_count * sizeof(uint3)));
			CCE(hipMemcpy(model_data->d_indices, model_data->h_indices, model_data->index_count * sizeof(uint3), hipMemcpyHostToDevice));

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_normals), model_data->vertex_count * sizeof(float3)));
			CCE(hipMemcpy(model_data->d_normals, model_data->h_normals, model_data->vertex_count * sizeof(float3), hipMemcpyHostToDevice));

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_uv), model_data->vertex_count * sizeof(float2)));
			CCE(hipMemcpy(model_data->d_uv, model_data->h_uv, model_data->vertex_count * sizeof(float2), hipMemcpyHostToDevice));
		}
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&d_texture_data_), texture_count * sizeof(Texture)));
	CCE(hipMalloc(reinterpret_cast<void**>(&d_material_data_), material_count * sizeof(Material)));
	CCE(hipMalloc(reinterpret_cast<void**>(&d_object_data_), object_count * sizeof(Object)));
	CCE(hipMemcpy(d_texture_data_, texture_data, texture_count * sizeof(Texture), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_material_data_, material_data, material_count * sizeof(Material), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_object_data_, object_data, object_count * sizeof(Object), hipMemcpyHostToDevice));

	const auto world = World(
		d_object_data_, 
		d_material_data_, 
		d_texture_data_, 
		static_cast<int32_t>(object_count),
		static_cast<int32_t>(material_count),
		static_cast<int32_t>(texture_count));

	CCE(hipMalloc(reinterpret_cast<void**>(&world_), sizeof(World)));
	CCE(hipMemcpy(world_, &world, sizeof(World), hipMemcpyHostToDevice));
}

void CudaRenderer::deallocate_world() const
{
	CCE(hipFree(world_));

	CCE(hipFree(d_object_data_));
	CCE(hipFree(d_material_data_));
	CCE(hipFree(d_texture_data_));

	for (const auto& object : world_info_->objects_)
	{
		if (object.type == ObjectType::MODEL)
		{
			CCE(hipFree(object.model.d_vertices));
			CCE(hipFree(object.model.d_indices));
			CCE(hipFree(object.model.d_normals));
			CCE(hipFree(object.model.d_uv));
		}
	}

	for (const auto& texture : world_info_->textures_)
	{
		if (texture.type == TextureType::IMAGE)
			CCE(hipFree(texture.image.d_data));
	}
}
