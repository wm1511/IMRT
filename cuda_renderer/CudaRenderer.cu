#include "hip/hip_runtime.h"
#include "CudaRenderer.cuh"
#include "Material.cuh"

#include <>
#include <cfloat>
#include <cstdio>

#define CCE(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__host__ void check_cuda(const hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
	    fprintf_s(stderr, "CUDA error = %u at %s: %i '%s' \n", (int32_t)result, file, line, func);
        hipDeviceReset();
        abort();
    }
}

__device__ float3 calculate_color(const Ray& ray, World** world, const float3* hdr_data, const RenderInfo render_info, uint32_t* random_state)
{
	Ray current_ray = ray;
    float3 current_absorption = make_float3(1.0f);
    const int32_t max_depth = render_info.max_depth;

    for (int32_t i = 0; i < max_depth; i++)
    {
	    Intersection intersection{};
        if ((*world)->intersect(current_ray, 0.001f, FLT_MAX, intersection))
        {
            Ray scattered_ray({0.0f, 0.0f, 0.0f}, {0.0f, 0.0f, 0.0f});
            float3 absorption;
            if (intersection.material->scatter(current_ray, intersection, absorption, scattered_ray, random_state))
            {
	            current_absorption *= absorption;
                current_ray = scattered_ray;
            }
            else return make_float3(0.0f);
        }
        else
        {
            if (hdr_data)
            {
                const float3 ray_direction = normalize(current_ray.direction());
                const float longitude = atan2(ray_direction.z, ray_direction.x);
	            const float latitude = acos(ray_direction.y);

			    const float u = longitude * kInv2Pi;
			    const float v = latitude * kInvPi;

	            const int32_t x = (int32_t)(u * (float)render_info.hdr_width);
	            const int32_t y = (int32_t)(v * (float)render_info.hdr_height);

	            const int32_t hdr_texel_index = x + y * render_info.hdr_width;
	            const float3 hdr_color = clamp(hdr_data[hdr_texel_index], 0.0f, 1.0f);
				return current_absorption * render_info.hdr_exposure * hdr_color; 
            }

            const float t = 0.5f * (versor(current_ray.direction()).y + 1.0f);
			const float3 color = (1.0f - t) * make_float3(1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
			return current_absorption * color;
        }
    }
	return make_float3(0.0f);
}

__global__ void render_pixel(float4* frame_buffer, float4* accumulation_buffer, Camera** camera, World** world, const float3* hdr_data, const RenderInfo render_info, uint4* xoshiro_state)
{
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t max_x = render_info.width;
	const uint32_t max_y = render_info.height;
    if (i >= max_x || j >= max_y) return;
	const uint32_t pixel_index = j * max_x + i;
    uint32_t local_random_state = xoshiro(&xoshiro_state[pixel_index]);

	const float u = ((float)i + pcg(&local_random_state)) / (float)max_x;
	const float v = ((float)j + pcg(&local_random_state)) / (float)max_y;
	const Ray ray = (*camera)->cast_ray(&local_random_state, u, v);
    const float3 color = sqrt(calculate_color(ray, world, hdr_data, render_info, &local_random_state));

    if (render_info.render_mode == PROGRESSIVE)
    {
    	accumulation_buffer[pixel_index] += make_float4(color, 1.0f);
    	frame_buffer[pixel_index] = accumulation_buffer[pixel_index] / (float)render_info.frames_since_refresh;
    }
    else if (render_info.render_mode == STATIC)
    {
        frame_buffer[pixel_index] += make_float4(color, 1.0f) / (float)render_info.samples_per_pixel;
    }
}

__global__ void random_init(const uint32_t max_x, const uint32_t max_y, uint4* xoshiro_state)
{
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= max_x || j >= max_y) return;
	const uint32_t pixel_index = j * max_x + i;
    xoshiro_state[pixel_index] = make_uint4(
        pixel_index + 15072003,
        pixel_index + 15112001,
        pixel_index + 10021151,
        pixel_index + 30027051);
}

__global__ void update_world(const RenderInfo render_info, MaterialInfo** material_data, Material** materials_list, ObjectInfo** object_data, Primitive** primitives_list)
{
	 if (threadIdx.x == 0 && blockIdx.x == 0) 
     {
         for (int32_t i = 0; i < render_info.material_count; i++)
         {
            if (material_data[i]->type == DIFFUSE)
            	materials_list[i] = new Diffuse((DiffuseInfo*)material_data[i]);
            else if (material_data[i]->type == SPECULAR)
            	materials_list[i] = new Specular((SpecularInfo*)material_data[i]);
            else if (material_data[i]->type == REFRACTIVE)
				materials_list[i] = new Refractive((RefractiveInfo*)material_data[i]);
         }

        for (int32_t i = 0; i < render_info.object_count; i++)
         {
            if (object_data[i]->type == SPHERE)
            	primitives_list[i] = new Sphere((SphereInfo*)object_data[i], materials_list[object_data[i]->material_id]);
            else if (object_data[i]->type == TRIANGLE)
            	primitives_list[i] = new Triangle((TriangleInfo*)object_data[i], materials_list[object_data[i]->material_id]);
         }
     }
}

__global__ void update_camera(Camera** camera, const RenderInfo render_info)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
		(*camera)->update(
			render_info.camera_position,
	        render_info.camera_direction,
	        render_info.fov,
			render_info.aperture,
	        render_info.focus_distance);
    }
}

__global__ void create_world(const RenderInfo render_info, MaterialInfo** material_data, ObjectInfo** object_data, Material** materials_list, Primitive** primitives_list, World** world)
{
	 if (threadIdx.x == 0 && blockIdx.x == 0) 
     {
         for (int32_t i = 0; i < render_info.material_count; i++)
         {
            if (material_data[i]->type == DIFFUSE)
            	materials_list[i] = new Diffuse((DiffuseInfo*)material_data[i]);
            else if (material_data[i]->type == SPECULAR)
            	materials_list[i] = new Specular((SpecularInfo*)material_data[i]);
            else if (material_data[i]->type == REFRACTIVE)
				materials_list[i] = new Refractive((RefractiveInfo*)material_data[i]);
         }

         for (int32_t i = 0; i < render_info.object_count; i++)
         {
            if (object_data[i]->type == SPHERE)
            	primitives_list[i] = new Sphere((SphereInfo*)object_data[i], materials_list[object_data[i]->material_id]);
            else if (object_data[i]->type == TRIANGLE)
            	primitives_list[i] = new Triangle((TriangleInfo*)object_data[i], materials_list[object_data[i]->material_id]);
         }

        *world = new World(primitives_list, render_info.object_count);
     }
}

__global__ void create_camera(Camera** camera, const RenderInfo render_info)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
		*camera = new Camera(
	            render_info.camera_position,
	            render_info.camera_direction,
	            render_info.fov,
	            (float)render_info.width / (float)render_info.height,
	            render_info.aperture,
	            render_info.focus_distance);
    }
}

__global__ void delete_world(Material** materials_list, Primitive** primitives_list, World** world, const uint32_t material_count, const uint32_t primitive_count)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
	    for (uint32_t i = 0; i < material_count; i++)
	    	delete materials_list[i];

    	for (uint32_t i = 0; i < primitive_count; i++)
    		delete primitives_list[i];
    	
    	delete *world;
    }
}

__global__ void delete_camera(Camera** camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
	    delete *camera;
}

CudaRenderer::CudaRenderer(const RenderInfo* render_info) : render_info_(render_info)
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;
    constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
    blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
    threads_ = dim3(thread_x, thread_y);

    CCE(hipMalloc((void**)&frame_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&accumulation_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&xoshiro_state_, sizeof(uint4) * width * height));

    random_init<<<blocks_, threads_>>>(width, height, xoshiro_state_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
    
    allocate_world();
    
    CCE(hipMalloc((void**)&camera_, sizeof(Camera*)));
    create_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());

    if (render_info_->hdr_data)
    {
        const uint64_t hdr_size = sizeof(float3) * render_info_->hdr_width * render_info_->hdr_height;
	    CCE(hipMalloc((void**)&device_hdr_data_, hdr_size));
    	CCE(hipMemcpy(device_hdr_data_, render_info_->hdr_data, hdr_size, hipMemcpyHostToDevice));
    }
}

CudaRenderer::~CudaRenderer()
{
    if (render_info_->hdr_data)
	    CCE(hipFree(device_hdr_data_));

    CCE(hipDeviceSynchronize());
    delete_camera<<<1, 1>>>(camera_);
    CCE(hipGetLastError());
    CCE(hipFree(camera_));

    deallocate_world();

    CCE(hipFree(xoshiro_state_));
    CCE(hipFree(accumulation_buffer_));
    CCE(hipFree(frame_buffer_));
    hipDeviceReset();
}

void CudaRenderer::render(float* image_data)
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;

    if (render_info_->render_mode == PROGRESSIVE)
	    render_pixel<<<blocks_, threads_>>>(frame_buffer_, accumulation_buffer_, camera_, world_, device_hdr_data_, *render_info_, xoshiro_state_);
    else if (render_info_->render_mode == STATIC)
	    for (int32_t i = 0; i < render_info_->samples_per_pixel; i++)
			render_pixel<<<blocks_, threads_>>>(frame_buffer_, accumulation_buffer_, camera_, world_, device_hdr_data_, *render_info_, xoshiro_state_);

	CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());

    CCE(hipMemcpy(image_data, frame_buffer_, sizeof(float4) * width * height, hipMemcpyDeviceToHost));
}

void CudaRenderer::refresh_buffer()
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;

    CCE(hipMemset(accumulation_buffer_, 0, sizeof(float4) * width * height));
    random_init<<<blocks_, threads_>>>(width, height, xoshiro_state_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_camera()
{
    update_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_world()
{
    reload_world();

    update_world<<<1, 1>>>(*render_info_, device_material_data_, materials_list_, device_object_data_, primitives_list_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_camera()
{
    CCE(hipDeviceSynchronize());
    delete_camera<<<1, 1>>>(camera_);
    CCE(hipGetLastError());

    create_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_image()
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
    blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
    threads_ = dim3(thread_x, thread_y);

    CCE(hipFree(frame_buffer_));
    CCE(hipFree(xoshiro_state_));
    CCE(hipFree(accumulation_buffer_));
    CCE(hipMalloc((void**)&accumulation_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&xoshiro_state_, sizeof(uint4) * width * height));
    CCE(hipMalloc((void**)&frame_buffer_, sizeof(float4) * width * height));
}

void CudaRenderer::recreate_world()
{
    deallocate_world();
    allocate_world();
}

void CudaRenderer::recreate_sky()
{
    CCE(hipFree(device_hdr_data_));

    if (render_info_->hdr_data)
    {
	    const uint64_t hdr_size = sizeof(float3) * render_info_->hdr_width * render_info_->hdr_height;
    	CCE(hipMalloc((void**)&device_hdr_data_, hdr_size));
    	CCE(hipMemcpy(device_hdr_data_, render_info_->hdr_data, hdr_size, hipMemcpyHostToDevice));
    }
    else
        device_hdr_data_ = nullptr;
}

void CudaRenderer::allocate_world()
{
    MaterialInfo** material_data = render_info_->material_data;
    ObjectInfo** object_data = render_info_->object_data;
    host_material_data_ = new MaterialInfo*[render_info_->material_data_count];
    host_object_data_ = new ObjectInfo*[render_info_->object_data_count];

    for (int32_t i = 0; i < render_info_->material_data_count; i++)
    {
        if (material_data[i]->type == DIFFUSE)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(DiffuseInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(DiffuseInfo), hipMemcpyHostToDevice));
        }
        else if (material_data[i]->type == SPECULAR)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(SpecularInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(SpecularInfo), hipMemcpyHostToDevice));
        }
        else if (material_data[i]->type == REFRACTIVE)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(RefractiveInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(RefractiveInfo), hipMemcpyHostToDevice));
        }
    }
    for (int32_t i = 0; i < render_info_->object_data_count; i++)
    {
        if (object_data[i]->type == SPHERE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(SphereInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(SphereInfo), hipMemcpyHostToDevice));
        }
        else if (object_data[i]->type == TRIANGLE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(TriangleInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(TriangleInfo), hipMemcpyHostToDevice));
        }
    }

    CCE(hipMalloc((void**)&device_material_data_, render_info_->material_data_count * sizeof(MaterialInfo*)));
    CCE(hipMalloc((void**)&device_object_data_, render_info_->object_data_count * sizeof(ObjectInfo*)));
	CCE(hipMemcpy(device_material_data_, host_material_data_, render_info_->material_data_count * sizeof(MaterialInfo*), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_object_data_, host_object_data_, render_info_->object_data_count * sizeof(ObjectInfo*), hipMemcpyHostToDevice));

    CCE(hipMalloc((void**)&primitives_list_, render_info_->object_data_count * sizeof(Primitive*)));
    CCE(hipMalloc((void**)&materials_list_, render_info_->material_data_count * sizeof(Material*)));
    CCE(hipMalloc((void**)&world_, sizeof(World*)));

    create_world<<<1, 1>>>(*render_info_, device_material_data_, device_object_data_, materials_list_, primitives_list_, world_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::deallocate_world() const
{
    CCE(hipDeviceSynchronize());
    //delete_world<<<1, 1>>>(materials_list_, primitives_list_, world_, render_info_->material_count, render_info_->object_count);
    CCE(hipGetLastError());
    CCE(hipFree(world_));

    CCE(hipFree(primitives_list_));
    CCE(hipFree(materials_list_));

    for (int32_t i = 0; i < render_info_->object_count; i++)
	    CCE(hipFree(host_object_data_[i]));
    for (int32_t i = 0; i < render_info_->material_count; i++)
		CCE(hipFree(host_material_data_[i]));
    CCE(hipFree(device_object_data_));
    CCE(hipFree(device_material_data_));

    delete[] host_object_data_;
    delete[] host_material_data_;
}

void CudaRenderer::reload_world() const
{
    MaterialInfo** material_data = render_info_->material_data;
    ObjectInfo** object_data = render_info_->object_data;

    for (int32_t i = 0; i < render_info_->material_data_count; i++)
    {
        if (material_data[i]->type == DIFFUSE)
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(DiffuseInfo), hipMemcpyHostToDevice));
        else if (material_data[i]->type == SPECULAR)
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(SpecularInfo), hipMemcpyHostToDevice));
        else if (material_data[i]->type == REFRACTIVE)
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(RefractiveInfo), hipMemcpyHostToDevice));
    }

    for (int32_t i = 0; i < render_info_->object_data_count; i++)
    {
        if (object_data[i]->type == SPHERE)
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(SphereInfo), hipMemcpyHostToDevice));
        else if (object_data[i]->type == TRIANGLE)
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(TriangleInfo), hipMemcpyHostToDevice));
    }

	CCE(hipMemcpy(device_material_data_, host_material_data_, render_info_->material_data_count * sizeof(MaterialInfo*), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_object_data_, host_object_data_, render_info_->object_data_count * sizeof(ObjectInfo*), hipMemcpyHostToDevice));
}
