#include "hip/hip_runtime.h"
//#include "stdafx.h"
#include "CudaRenderer.cuh"

#include <>

#include <cstdio>

#define CCE(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__host__ void check_cuda(const hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
	    printf("CUDA error = %u at %s: %i '%s' \n", (int32_t)result, file, line, func);
        hipDeviceReset();
        abort();
    }
}

__global__ void render_pixel(float4* frame_buffer, float4* accumulation_buffer, Camera** camera, World** world, const float3* hdr_data, const RenderInfo render_info, uint4* xoshiro_state)
{
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t max_x = render_info.width;
	const uint32_t max_y = render_info.height;
    if (i >= max_x || j >= max_y) return;
	const uint32_t pixel_index = j * max_x + i;
    uint32_t local_random_state = xoshiro(&xoshiro_state[pixel_index]);

	const float u = ((float)i + pcg(&local_random_state)) / (float)max_x;
	const float v = ((float)j + pcg(&local_random_state)) / (float)max_y;
	const Ray ray = (*camera)->cast_ray(&local_random_state, u, v);
    const float3 color = sqrt(calculate_color(ray, world, hdr_data, render_info, &local_random_state));

    if (render_info.render_mode == PROGRESSIVE)
    {
    	accumulation_buffer[pixel_index] += make_float4(color, 1.0f);
    	frame_buffer[pixel_index] = accumulation_buffer[pixel_index] / (float)render_info.frames_since_refresh;
    }
    else if (render_info.render_mode == STATIC)
    {
        frame_buffer[pixel_index] += make_float4(color, 1.0f) / (float)render_info.samples_per_pixel;
    }
}

__global__ void random_init(const uint32_t max_x, const uint32_t max_y, uint4* xoshiro_state)
{
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= max_x || j >= max_y) return;
	const uint32_t pixel_index = j * max_x + i;
    xoshiro_state[pixel_index] = make_uint4(
        pixel_index + 15072003,
        pixel_index + 15112001,
        pixel_index + 10021151,
        pixel_index + 30027051);
}

__global__ void update_camera(Camera** camera, const RenderInfo render_info)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
		(*camera)->update(
			render_info.camera_position,
	        render_info.camera_direction,
	        render_info.fov,
			render_info.aperture,
	        render_info.focus_distance);
}

__global__ void update_material(World** world, const int32_t index, MaterialInfo** material_data)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
		(*world)->update_material(index, material_data[index]);
}

__global__ void update_object(World** world, const int32_t index, ObjectInfo** object_data)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
		(*world)->update_object(index, object_data[index]);
}

__global__ void create_world(ObjectInfo** object_data, MaterialInfo** material_data, const int32_t object_count, const int32_t material_count, World** world)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
       *world = new World(object_data, material_data, object_count, material_count);
}

__global__ void create_camera(Camera** camera, const RenderInfo render_info)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
		*camera = new Camera(
	            render_info.camera_position,
	            render_info.camera_direction,
	            render_info.fov,
	            (float)render_info.width / (float)render_info.height,
	            render_info.aperture,
	            render_info.focus_distance);
}

__global__ void delete_world(World** world)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    	delete *world;
}

__global__ void delete_camera(Camera** camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
	    delete *camera;
}

CudaRenderer::CudaRenderer(const RenderInfo* render_info, const WorldInfo* world_info) : render_info_(render_info), world_info_(world_info)
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;
    constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
    blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
    threads_ = dim3(thread_x, thread_y);

    CCE(hipMalloc((void**)&frame_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&accumulation_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&xoshiro_state_, sizeof(uint4) * width * height));

    random_init<<<blocks_, threads_>>>(width, height, xoshiro_state_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
    
    allocate_world();
    
    CCE(hipMalloc((void**)&camera_, sizeof(Camera*)));
    create_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());

    if (render_info_->hdr_data)
    {
        const uint64_t hdr_size = sizeof(float3) * render_info_->hdr_width * render_info_->hdr_height;
	    CCE(hipMalloc((void**)&device_hdr_data_, hdr_size));
    	CCE(hipMemcpy(device_hdr_data_, render_info_->hdr_data, hdr_size, hipMemcpyHostToDevice));
    }
}

CudaRenderer::~CudaRenderer()
{
    if (render_info_->hdr_data)
	    CCE(hipFree(device_hdr_data_));

    CCE(hipDeviceSynchronize());
    delete_camera<<<1, 1>>>(camera_);
    CCE(hipGetLastError());
    CCE(hipFree(camera_));

    deallocate_world();

    CCE(hipFree(xoshiro_state_));
    CCE(hipFree(accumulation_buffer_));
    CCE(hipFree(frame_buffer_));
    hipDeviceReset();
}

void CudaRenderer::render(float* image_data)
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;

    if (render_info_->render_mode == PROGRESSIVE)
	    render_pixel<<<blocks_, threads_>>>(frame_buffer_, accumulation_buffer_, camera_, world_, device_hdr_data_, *render_info_, xoshiro_state_);
    else if (render_info_->render_mode == STATIC)
	    for (int32_t i = 0; i < render_info_->samples_per_pixel; i++)
			render_pixel<<<blocks_, threads_>>>(frame_buffer_, accumulation_buffer_, camera_, world_, device_hdr_data_, *render_info_, xoshiro_state_);

	CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());

    if (render_info_->frame_needs_display)
		CCE(hipMemcpy(image_data, frame_buffer_, sizeof(float4) * width * height, hipMemcpyDeviceToHost));
}

void CudaRenderer::refresh_buffer()
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;

    CCE(hipMemset(accumulation_buffer_, 0, sizeof(float4) * width * height));
    random_init<<<blocks_, threads_>>>(width, height, xoshiro_state_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_camera()
{
    update_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_material(const int32_t index) const
{
	const MaterialInfo* material = world_info_->materials_[index];

    if (material->type == DIFFUSE)
		CCE(hipMemcpy(host_material_data_[index], material, sizeof(DiffuseInfo), hipMemcpyHostToDevice));
    else if (material->type == SPECULAR)
		CCE(hipMemcpy(host_material_data_[index], material, sizeof(SpecularInfo), hipMemcpyHostToDevice));
    else if (material->type == REFRACTIVE)
		CCE(hipMemcpy(host_material_data_[index], material, sizeof(RefractiveInfo), hipMemcpyHostToDevice));

    CCE(hipMemcpy(&device_material_data_[index], &host_material_data_[index], sizeof(MaterialInfo*), hipMemcpyHostToDevice));

    update_material<<<1, 1>>>(world_, index, device_material_data_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_object(const int32_t index) const
{
	ObjectInfo* object = world_info_->objects_[index];

    if (object->type == SPHERE)
		CCE(hipMemcpy(host_object_data_[index], object, sizeof(SphereInfo), hipMemcpyHostToDevice));
    else if (object->type == TRIANGLE)
		CCE(hipMemcpy(host_object_data_[index], object, sizeof(TriangleInfo), hipMemcpyHostToDevice));
	else if (object->type == PLANE)
		CCE(hipMemcpy(host_object_data_[index], object, sizeof(PlaneInfo), hipMemcpyHostToDevice));
    else if (object->type == MODEL)
    {
        TriangleInfo* triangle_data = ((ModelInfo*)object)->triangles;
        ((ModelInfo*)object)->triangles = host_triangle_data_[index]; 
	    CCE(hipMemcpy(host_object_data_[index], object, sizeof(ModelInfo), hipMemcpyHostToDevice));
        ((ModelInfo*)world_info_->objects_[index])->triangles = triangle_data;
    }

    CCE(hipMemcpy(&device_object_data_[index], &host_object_data_[index], sizeof(ObjectInfo*), hipMemcpyHostToDevice));

    update_object<<<1, 1>>>(world_, index, device_object_data_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_camera()
{
    CCE(hipDeviceSynchronize());
    delete_camera<<<1, 1>>>(camera_);
    CCE(hipGetLastError());

    create_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_image()
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
    blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
    threads_ = dim3(thread_x, thread_y);

    CCE(hipFree(frame_buffer_));
    CCE(hipFree(xoshiro_state_));
    CCE(hipFree(accumulation_buffer_));
    CCE(hipMalloc((void**)&accumulation_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&xoshiro_state_, sizeof(uint4) * width * height));
    CCE(hipMalloc((void**)&frame_buffer_, sizeof(float4) * width * height));
}

void CudaRenderer::recreate_sky()
{
    CCE(hipFree(device_hdr_data_));

    if (render_info_->hdr_data)
    {
	    const uint64_t hdr_size = sizeof(float3) * render_info_->hdr_width * render_info_->hdr_height;
    	CCE(hipMalloc((void**)&device_hdr_data_, hdr_size));
    	CCE(hipMemcpy(device_hdr_data_, render_info_->hdr_data, hdr_size, hipMemcpyHostToDevice));
    }
    else
        device_hdr_data_ = nullptr;
}

void CudaRenderer::allocate_world()
{
	const std::vector<MaterialInfo*> material_data = world_info_->materials_;
	const std::vector<ObjectInfo*> object_data = world_info_->objects_;
    const auto material_count = material_data.size();
    const auto object_count = object_data.size();
    host_material_data_ = new MaterialInfo*[material_count];
    host_object_data_ = new ObjectInfo*[object_count];
    host_triangle_data_ = new TriangleInfo*[object_count];

    for (uint64_t i = 0; i < material_count; i++)
    {
        if (material_data[i]->type == DIFFUSE)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(DiffuseInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(DiffuseInfo), hipMemcpyHostToDevice));
        }
        else if (material_data[i]->type == SPECULAR)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(SpecularInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(SpecularInfo), hipMemcpyHostToDevice));
        }
        else if (material_data[i]->type == REFRACTIVE)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(RefractiveInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(RefractiveInfo), hipMemcpyHostToDevice));
        }
    }
    for (uint64_t i = 0; i < object_count; i++)
    {
        if (object_data[i]->type == SPHERE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(SphereInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(SphereInfo), hipMemcpyHostToDevice));
        }
        else if (object_data[i]->type == TRIANGLE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(TriangleInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(TriangleInfo), hipMemcpyHostToDevice));
        }
    	else if (object_data[i]->type == PLANE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(PlaneInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(PlaneInfo), hipMemcpyHostToDevice));
        }
        else if (object_data[i]->type == MODEL)
        {
	        const auto model_data = (ModelInfo*)object_data[i];
            TriangleInfo* triangle_data = model_data->triangles;

            CCE(hipMalloc((void**)&host_triangle_data_[i], model_data->triangle_count * sizeof(TriangleInfo)));
			CCE(hipMemcpy(host_triangle_data_[i], model_data->triangles, model_data->triangle_count * sizeof(TriangleInfo), hipMemcpyHostToDevice));
            model_data->triangles = host_triangle_data_[i];

            CCE(hipMalloc((void**)&host_object_data_[i], sizeof(ModelInfo)));
			CCE(hipMemcpy(host_object_data_[i], model_data, sizeof(ModelInfo), hipMemcpyHostToDevice));
            ((ModelInfo*)world_info_->objects_[i])->triangles = triangle_data;
        }
    }

    CCE(hipMalloc((void**)&device_material_data_, material_count * sizeof(MaterialInfo*)));
    CCE(hipMalloc((void**)&device_object_data_, object_count * sizeof(ObjectInfo*)));
	CCE(hipMemcpy(device_material_data_, host_material_data_, material_count * sizeof(MaterialInfo*), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_object_data_, host_object_data_, object_count * sizeof(ObjectInfo*), hipMemcpyHostToDevice));

    CCE(hipMalloc((void**)&world_, sizeof(World*)));
    create_world<<<1, 1>>>(device_object_data_, device_material_data_, (int32_t)object_count, (int32_t)material_count, world_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::deallocate_world() const
{
    CCE(hipDeviceSynchronize());
	//delete_world<<<1, 1>>>(world_);
    CCE(hipGetLastError());
    CCE(hipFree(world_));

    CCE(hipFree(device_object_data_));
    CCE(hipFree(device_material_data_));

    for (uint64_t i = 0; i < world_info_->objects_.size(); i++)
    {
        if (world_info_->objects_[i]->type == MODEL)
            CCE(hipFree(host_triangle_data_[i]));

	    CCE(hipFree(host_object_data_[i]));
    }
    
    for (uint64_t i = 0; i < world_info_->materials_.size(); i++)
		CCE(hipFree(host_material_data_[i]));

    delete[] host_triangle_data_;
    delete[] host_object_data_;
    delete[] host_material_data_;
}
