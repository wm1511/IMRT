#include "hip/hip_runtime.h"
#include "CudaRenderer.cuh"
#include "Material.cuh"

#include <>
#include <cfloat>
#include <cstdio>

#define CCE(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__host__ void check_cuda(const hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
	    fprintf_s(stderr, "CUDA error = %u at %s: %i '%s' \n", (int32_t)result, file, line, func);
        hipDeviceReset();
        abort();
    }
}

__device__ float3 calculate_color(const Ray& ray, World** world, const RenderInfo render_info, uint32_t* random_state)
{
	Ray current_ray = ray;
    float3 current_absorption = make_float3(1.0f);
    const int32_t max_depth = render_info.max_depth;

    for (int32_t i = 0; i < max_depth; i++)
    {
	    Intersection intersection{};
        if ((*world)->intersect(current_ray, 0.001f, FLT_MAX, intersection))
        {
            Ray scattered_ray({0.0f, 0.0f, 0.0f}, {0.0f, 0.0f, 0.0f});
            float3 absorption;
            if (intersection.material->scatter(current_ray, intersection, absorption, scattered_ray, random_state))
            {
	            current_absorption *= absorption;
                current_ray = scattered_ray;
            }
            else return make_float3(0.0f);
        }
        else
        {
	        const float t = 0.5f * (versor(current_ray.direction()).y + 1.0f);
	        const float3 color = (1.0f - t) * make_float3(1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
            return current_absorption * color;
        }
    }
	return make_float3(0.0f);
}

__global__ void render_init(const uint32_t max_x, const uint32_t max_y, uint32_t* random_state)
{
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= max_x || j >= max_y) return;
	const uint32_t pixel_index = j * max_x + i;
    random_state[pixel_index] = 1511 + pixel_index;
}

__global__ void render_pixel(float4* frame_buffer, float4* accumulation_buffer, const uint32_t max_x, const uint32_t max_y, Camera** camera, World** world, const RenderInfo render_info, uint32_t* random_state)
{
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= max_x || j >= max_y) return;
	const uint32_t pixel_index = j * max_x + i;
    uint32_t local_random_state = random_state[pixel_index];
    //const int32_t samples_per_pixel = render_info.samples_per_pixel;

    /*float3 color = make_float3(0.0f);
    for (int32_t s = 0; s < samples_per_pixel; s++)
    {
	    const float u = ((float)i + pcg_rxs_m_xs(&local_random_state)) / (float)max_x;
	    const float v = ((float)j + pcg_rxs_m_xs(&local_random_state)) / (float)max_y;
        Ray ray = (*camera)->cast_ray(&local_random_state, u, v);
        color += calculate_color(ray, world, render_info, &local_random_state);
    }*/
    const float u = ((float)i + pcg_rxs_m_xs(&local_random_state)) / (float)max_x;
	const float v = ((float)j + pcg_rxs_m_xs(&local_random_state)) / (float)max_y;
	const Ray ray = (*camera)->cast_ray(&local_random_state, u, v);
	const float3 color = sqrt(calculate_color(ray, world, render_info, &local_random_state));
    //random_state[pixel_index] = local_random_state;
    //color /= (float)samples_per_pixel;
    //color = sqrt(color);
    accumulation_buffer[pixel_index] += make_float4(color, 1.0f);
    frame_buffer[pixel_index] = accumulation_buffer[pixel_index] / (float)render_info.frames_since_refresh;
}

__global__ void create_world(const RenderInfo render_info, MaterialInfo** material_data, ObjectInfo** object_data, Material** materials_list, Primitive** primitives_list, World** world)
{
	 if (threadIdx.x == 0 && blockIdx.x == 0) 
     {
         for (int32_t i = 0; i < render_info.material_count; i++)
         {
            if (material_data[i]->type == DIFFUSE)
            	materials_list[i] = new Diffuse((DiffuseInfo*)material_data[i]);
            else if (material_data[i]->type == SPECULAR)
            	materials_list[i] = new Specular((SpecularInfo*)material_data[i]);
            else if (material_data[i]->type == REFRACTIVE)
				materials_list[i] = new Refractive((RefractiveInfo*)material_data[i]);
         }

         for (int32_t i = 0; i < render_info.object_count; i++)
         {
            if (object_data[i]->type == SPHERE)
            	primitives_list[i] = new Sphere((SphereInfo*)object_data[i], materials_list[object_data[i]->material_id]);
            else if (object_data[i]->type == TRIANGLE)
            	primitives_list[i] = new Triangle((TriangleInfo*)object_data[i], materials_list[object_data[i]->material_id]);
         }

        *world = new World(primitives_list, render_info.object_count);
     }
}

__global__ void create_camera(Camera** camera, const RenderInfo render_info)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
		*camera = new Camera(
	            make_float3(render_info.look_origin),
	            make_float3(render_info.look_target),
	            render_info.fov,
	            (float)render_info.width / (float)render_info.height,
	            render_info.aperture,
	            render_info.focus_distance);
    }
}

__global__ void delete_world(Material** materials_list, Primitive** primitives_list, World** world, const uint32_t material_count, const uint32_t primitive_count)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
	    for (uint32_t i = 0; i < material_count; i++)
	    	delete materials_list[i];

    	for (uint32_t i = 0; i < primitive_count; i++)
    		delete primitives_list[i];
    	
    	delete *world;
    }
}

__global__ void delete_camera(Camera** camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
	    delete *camera;
}

CudaRenderer::CudaRenderer(const RenderInfo* render_info) : render_info_(render_info)
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;

    CCE(hipMalloc((void**)&frame_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&accumulation_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&random_state_, sizeof(uint32_t) * width * height));
    
    allocate_world();
    
    CCE(hipMalloc((void**)&camera_, sizeof(Camera*)));
    create_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

CudaRenderer::~CudaRenderer()
{
    CCE(hipDeviceSynchronize());
    delete_camera<<<1, 1>>>(camera_);
    CCE(hipGetLastError());
    CCE(hipFree(camera_));

    deallocate_world();

    CCE(hipFree(random_state_));
    CCE(hipFree(accumulation_buffer_));
    CCE(hipFree(frame_buffer_));
    hipDeviceReset();
}

void CudaRenderer::render(float* image_data)
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
    dim3 blocks((render_info_->width + thread_x - 1) / thread_x, (render_info_->height + thread_y - 1) / thread_y);
    dim3 threads(thread_x, thread_y);

    render_init<<<blocks, threads>>>(width, height, random_state_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
    render_pixel<<<blocks, threads>>>(frame_buffer_, accumulation_buffer_, width, height, camera_, world_, *render_info_, random_state_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());

    CCE(hipMemcpy(image_data, frame_buffer_, sizeof(float4) * width * height, hipMemcpyDeviceToHost));
}

void CudaRenderer::recreate_camera()
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;

    hipMemset(accumulation_buffer_, 0, sizeof(float4) * width * height);

    CCE(hipDeviceSynchronize());
    delete_camera<<<1, 1>>>(camera_);
    CCE(hipGetLastError());

    create_camera<<<1, 1>>>(camera_, *render_info_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_image()
{
    const uint32_t width = render_info_->width;
    const uint32_t height = render_info_->height;
    
    hipMemset(accumulation_buffer_, 0, sizeof(float4) * width * height);

    CCE(hipFree(random_state_));
    CCE(hipFree(frame_buffer_));
    CCE(hipMalloc((void**)&frame_buffer_, sizeof(float4) * width * height));
    CCE(hipMalloc((void**)&random_state_, sizeof(uint32_t) * width * height));
}

void CudaRenderer::recreate_world()
{
    hipMemset(accumulation_buffer_, 0, sizeof(float4) * render_info_->width * render_info_->height);

    deallocate_world();
    allocate_world();
}

void CudaRenderer::allocate_world()
{
    MaterialInfo** material_data = render_info_->material_data;
    ObjectInfo** object_data = render_info_->object_data;
    host_material_data_ = new MaterialInfo*[render_info_->material_data_count];
    host_object_data_ = new ObjectInfo*[render_info_->object_data_count];

    for (int32_t i = 0; i < render_info_->material_data_count; i++)
    {
        if (material_data[i]->type == DIFFUSE)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(DiffuseInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(DiffuseInfo), hipMemcpyHostToDevice));
        }
        else if (material_data[i]->type == SPECULAR)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(SpecularInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(SpecularInfo), hipMemcpyHostToDevice));
        }
        else if (material_data[i]->type == REFRACTIVE)
        {
	        CCE(hipMalloc((void**)&host_material_data_[i], sizeof(RefractiveInfo)));
			CCE(hipMemcpy(host_material_data_[i], material_data[i], sizeof(RefractiveInfo), hipMemcpyHostToDevice));
        }
    }
    for (int32_t i = 0; i < render_info_->object_data_count; i++)
    {
        if (object_data[i]->type == SPHERE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(SphereInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(SphereInfo), hipMemcpyHostToDevice));
        }
        else if (object_data[i]->type == TRIANGLE)
        {
	        CCE(hipMalloc((void**)&host_object_data_[i], sizeof(TriangleInfo)));
			CCE(hipMemcpy(host_object_data_[i], object_data[i], sizeof(TriangleInfo), hipMemcpyHostToDevice));
        }
    }

    CCE(hipMalloc((void**)&device_material_data_, render_info_->material_data_count * sizeof(MaterialInfo*)));
    CCE(hipMalloc((void**)&device_object_data_, render_info_->object_data_count * sizeof(ObjectInfo*)));
	CCE(hipMemcpy(device_material_data_, host_material_data_, render_info_->material_data_count * sizeof(MaterialInfo*), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_object_data_, host_object_data_, render_info_->object_data_count * sizeof(ObjectInfo*), hipMemcpyHostToDevice));

    CCE(hipMalloc((void**)&primitives_list_, render_info_->object_data_count * sizeof(Primitive*)));
    CCE(hipMalloc((void**)&materials_list_, render_info_->material_data_count * sizeof(Material*)));
    CCE(hipMalloc((void**)&world_, sizeof(World*)));

    create_world<<<1, 1>>>(*render_info_, device_material_data_, device_object_data_, materials_list_, primitives_list_, world_);
    CCE(hipGetLastError());
    CCE(hipDeviceSynchronize());
}

void CudaRenderer::deallocate_world() const
{
    CCE(hipDeviceSynchronize());
    //delete_world<<<1, 1>>>(materials_list_, primitives_list_, world_, render_info_->material_count, render_info_->object_count);
    CCE(hipGetLastError());
    CCE(hipFree(world_));

    CCE(hipFree(primitives_list_));
    CCE(hipFree(materials_list_));

    for (int32_t i = 0; i < render_info_->object_count; i++)
	    CCE(hipFree(host_object_data_[i]));
    for (int32_t i = 0; i < render_info_->material_count; i++)
		CCE(hipFree(host_material_data_[i]));
    CCE(hipFree(device_object_data_));
    CCE(hipFree(device_material_data_));

    delete[] host_object_data_;
    delete[] host_material_data_;
}
