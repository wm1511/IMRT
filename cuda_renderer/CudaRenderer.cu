#include "hip/hip_runtime.h"
// ReSharper disable once CppPrecompiledHeaderIsNotIncluded
#include "CudaRenderer.cuh"
#include "CudaKernels.cuh"

#include "../common/Utils.hpp"

#include <hip/hip_runtime.h>

CudaRenderer::CudaRenderer(const RenderInfo* render_info, const WorldInfo* world_info, const SkyInfo* sky_info)
	: render_info_(render_info), world_info_(world_info), sky_info_(sky_info)
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
	blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
	threads_ = dim3(thread_x, thread_y);

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	CCE(hipMalloc(reinterpret_cast<void**>(&accumulation_buffer_), sizeof(float4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_initial_), sizeof(uint4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_state_), sizeof(uint4) * width * height));

	random_init<<<blocks_, threads_>>>(width, height, xoshiro_initial_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	CCE(hipMemcpy(xoshiro_state_, xoshiro_initial_, sizeof(uint4) * width * height, hipMemcpyDeviceToDevice));

	allocate_world();

	CCE(hipMalloc(reinterpret_cast<void**>(&camera_), sizeof(hipDeviceptr_t)));
	create_camera<<<1, 1>>>(camera_, *render_info_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	if (sky_info_->buffered_hdr_data)
	{
		const uint64_t hdr_size = sizeof(float3) * sky_info_->hdr_width * sky_info_->hdr_height;
		CCE(hipMalloc(reinterpret_cast<void**>(&sky_info_->usable_hdr_data), hdr_size));
		CCE(hipMemcpy(sky_info_->usable_hdr_data, sky_info_->buffered_hdr_data, hdr_size, hipMemcpyHostToDevice));
	}
}

CudaRenderer::~CudaRenderer()
{
	if (sky_info_->buffered_hdr_data)
		CCE(hipFree(sky_info_->usable_hdr_data));

	CCE(hipDeviceSynchronize());
	delete_camera << <1, 1 >> > (camera_);
	CCE(hipGetLastError());
	CCE(hipFree(camera_));

	deallocate_world();

	CCE(hipFree(xoshiro_state_));
	CCE(hipFree(xoshiro_initial_));
	CCE(hipFree(accumulation_buffer_));
	hipDeviceReset();
}

void CudaRenderer::render_static()
{
	frame_buffer_ = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	CCE(hipMemset(frame_buffer_, 0, render_info_->frame_size));

	for (int32_t i = 0; i < render_info_->samples_per_pixel; i++)
		render_pixel_static<<<blocks_, threads_>>>(frame_buffer_, camera_, world_, *sky_info_, *render_info_, xoshiro_state_);

	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	CCE(hipFree(frame_buffer_));
}

void CudaRenderer::render_progressive()
{
	frame_buffer_ = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	render_pixel_progressive<<<blocks_, threads_>>>(frame_buffer_, accumulation_buffer_, camera_, world_, *sky_info_, *render_info_, xoshiro_state_);

	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	CCE(hipFree(frame_buffer_));
}

void CudaRenderer::refresh_buffer()
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;

	CCE(hipMemset(accumulation_buffer_, 0, sizeof(float4) * width * height));
	CCE(hipMemcpy(xoshiro_state_, xoshiro_initial_, sizeof(uint4) * width * height, hipMemcpyDeviceToDevice));
}

void CudaRenderer::refresh_camera()
{
	update_camera<<<1, 1>>>(camera_, *render_info_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_texture(const int32_t index) const
{
	const TextureInfo* texture = world_info_->textures_[index];

	CCE(hipMemcpy(host_texture_data_[index], texture, texture->get_size(), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_texture_data_ + index, host_texture_data_ + index, sizeof(hipDeviceptr_t), hipMemcpyHostToDevice));

	update_texture<<<1, 1>>>(world_, index, device_texture_data_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_material(const int32_t index) const
{
	const MaterialInfo* material = world_info_->materials_[index];

	CCE(hipMemcpy(host_material_data_[index], material, material->get_size(), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_material_data_ + index, host_material_data_ + index, sizeof(hipDeviceptr_t), hipMemcpyHostToDevice));

	update_material<<<1, 1 >>>(world_, index, device_material_data_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::refresh_object(const int32_t index) const
{
	const ObjectInfo* object = world_info_->objects_[index];

	CCE(hipMemcpy(host_object_data_[index], object, object->get_size(), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_object_data_ + index, host_object_data_ + index, sizeof(hipDeviceptr_t), hipMemcpyHostToDevice));

	update_object<<<1, 1 >>>(world_, index, device_object_data_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_camera()
{
	CCE(hipDeviceSynchronize());
	delete_camera << <1, 1 >> > (camera_);
	CCE(hipGetLastError());

	create_camera<<<1, 1 >>>(camera_, *render_info_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_image()
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
	blocks_ = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
	threads_ = dim3(thread_x, thread_y);

	CCE(hipFree(xoshiro_state_));
	CCE(hipFree(xoshiro_initial_));
	CCE(hipFree(accumulation_buffer_));
	CCE(hipMalloc(reinterpret_cast<void**>(&accumulation_buffer_), sizeof(float4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_initial_), sizeof(uint4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_state_), sizeof(uint4) * width * height));

	random_init<<<blocks_, threads_>>>(width, height, xoshiro_initial_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::recreate_sky()
{
	CCE(hipFree(sky_info_->usable_hdr_data));

	if (sky_info_->buffered_hdr_data)
	{
		const uint64_t hdr_size = sizeof(float3) * sky_info_->hdr_width * sky_info_->hdr_height;
		CCE(hipMalloc(reinterpret_cast<void**>(&sky_info_->usable_hdr_data), hdr_size));
		CCE(hipMemcpy(sky_info_->usable_hdr_data, sky_info_->buffered_hdr_data, hdr_size, hipMemcpyHostToDevice));
	}
	else
		sky_info_->usable_hdr_data = nullptr;
}

void CudaRenderer::map_frame_memory()
{
	frame_buffer_ = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	CCE(hipMemcpy(render_info_->frame_data, frame_buffer_, render_info_->frame_size, hipMemcpyDeviceToHost));
	CCE(hipFree(frame_buffer_));
}

void CudaRenderer::allocate_world()
{
	const std::vector<TextureInfo*> texture_data = world_info_->textures_;
	const std::vector<MaterialInfo*> material_data = world_info_->materials_;
	const std::vector<ObjectInfo*> object_data = world_info_->objects_;
	const auto texture_count = texture_data.size();
	const auto material_count = material_data.size();
	const auto object_count = object_data.size();
	host_texture_data_ = new TextureInfo * [texture_count];
	host_material_data_ = new MaterialInfo * [material_count];
	host_object_data_ = new ObjectInfo * [object_count];

	for (uint64_t i = 0; i < texture_count; i++)
	{
		if (texture_data[i]->type == TextureType::IMAGE)
		{
			const auto image_data = dynamic_cast<ImageInfo*>(texture_data[i]);
			const uint64_t image_size = sizeof(float) * image_data->width * image_data->height * 3;

			CCE(hipMalloc(reinterpret_cast<void**>(&image_data->usable_data), image_size));
			CCE(hipMemcpy(image_data->usable_data, image_data->buffered_data, image_size, hipMemcpyHostToDevice));
		}

		CCE(hipMalloc(reinterpret_cast<void**>(&host_texture_data_[i]), texture_data[i]->get_size()));
		CCE(hipMemcpy(host_texture_data_[i], texture_data[i], texture_data[i]->get_size(), hipMemcpyHostToDevice));
	}

	for (uint64_t i = 0; i < material_count; i++)
	{
		CCE(hipMalloc(reinterpret_cast<void**>(&host_material_data_[i]), material_data[i]->get_size()));
		CCE(hipMemcpy(host_material_data_[i], material_data[i], material_data[i]->get_size(), hipMemcpyHostToDevice));
	}

	for (uint64_t i = 0; i < object_count; i++)
	{
		if (object_data[i]->type == ObjectType::MODEL)
		{
			const auto model_data = dynamic_cast<ModelInfo*>(object_data[i]);

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->usable_vertices), 3 * model_data->triangle_count * sizeof(Vertex)));
			CCE(hipMemcpy(model_data->usable_vertices, model_data->buffered_vertices, 3 * model_data->triangle_count * sizeof(Vertex), hipMemcpyHostToDevice));
		}

		CCE(hipMalloc(reinterpret_cast<void**>(&host_object_data_[i]), object_data[i]->get_size()));
		CCE(hipMemcpy(host_object_data_[i], object_data[i], object_data[i]->get_size(), hipMemcpyHostToDevice));
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&device_texture_data_), texture_count * sizeof(hipDeviceptr_t)));
	CCE(hipMalloc(reinterpret_cast<void**>(&device_material_data_), material_count * sizeof(hipDeviceptr_t)));
	CCE(hipMalloc(reinterpret_cast<void**>(&device_object_data_), object_count * sizeof(hipDeviceptr_t)));
	CCE(hipMemcpy(device_texture_data_, host_texture_data_, texture_count * sizeof(hipDeviceptr_t), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_material_data_, host_material_data_, material_count * sizeof(hipDeviceptr_t), hipMemcpyHostToDevice));
	CCE(hipMemcpy(device_object_data_, host_object_data_, object_count * sizeof(hipDeviceptr_t), hipMemcpyHostToDevice));

	CCE(hipMalloc(reinterpret_cast<void**>(&world_), sizeof(hipDeviceptr_t)));
	create_world<<<1, 1 >>>(
		device_object_data_,
		device_material_data_,
		device_texture_data_,
		static_cast<int32_t>(object_count),
		static_cast<int32_t>(material_count),
		static_cast<int32_t>(texture_count), world_);

	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void CudaRenderer::deallocate_world() const
{
	CCE(hipDeviceSynchronize());
	delete_world << <1, 1 >> > (world_);
	CCE(hipGetLastError());
	CCE(hipFree(world_));

	CCE(hipFree(device_object_data_));
	CCE(hipFree(device_material_data_));
	CCE(hipFree(device_texture_data_));

	for (uint64_t i = 0; i < world_info_->objects_.size(); i++)
	{
		if (world_info_->objects_[i]->type == ObjectType::MODEL)
			CCE(hipFree(dynamic_cast<ModelInfo*>(world_info_->objects_[i])->usable_vertices));

		CCE(hipFree(host_object_data_[i]));
	}

	for (uint64_t i = 0; i < world_info_->materials_.size(); i++)
	{
		CCE(hipFree(host_material_data_[i]));
	}

	for (uint64_t i = 0; i < world_info_->textures_.size(); i++)
	{
		if (world_info_->textures_[i]->type == TextureType::IMAGE)
			CCE(hipFree(dynamic_cast<ImageInfo*>(world_info_->textures_[i])->usable_data));

		CCE(hipFree(host_texture_data_[i]));
	}

	delete[] host_object_data_;
	delete[] host_material_data_;
	delete[] host_texture_data_;
}
