#include "CudaRenderer.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>

CudaRenderer::CudaRenderer(const RtInfo& rt_info) : rt_info_(rt_info)
{
}

void CudaRenderer::render(uint32_t* image_data, const uint32_t width, const uint32_t height)
{
    hiprandGenerator_t gen;
    float *dev_data;

    hipMalloc(reinterpret_cast<void**>(&dev_data), width * height * sizeof(float));

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateUniform(gen, dev_data, width * height);

    hipMemcpy(image_data, dev_data, width * height * sizeof(float), hipMemcpyDeviceToHost);

    hiprandDestroyGenerator(gen);
    hipFree(dev_data);
}