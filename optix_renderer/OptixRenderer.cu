#include "hip/hip_runtime.h"
// ReSharper disable once CppPrecompiledHeaderIsNotIncluded
#include "OptixRenderer.cuh"

#include "../common/Utils.hpp"
#include "../cuda_renderer/CudaKernels.cuh"

#include <optix_stubs.h>
#include <optix_function_table_definition.h>

#ifdef _DEBUG
static void context_log(unsigned int level, const char* tag, const char* message, void*)
{
	printf("[%u][%s]: %s\n", level, tag, message);
}
#endif

//static void prepare_sphere_input(std::vector<OptixBuildInput>& inputs, Sphere& sphere, uint32_t* flags, std::vector<float3*>& centers, std::vector<float*>& radii)
//{
//	OptixBuildInput sphere_input{};
//	sphere_input.type = OPTIX_BUILD_INPUT_TYPE_SPHERES;
//
//	centers.resize(centers.size() + 1);
//	radii.resize(radii.size() + 1);
//	auto& d_center = centers[centers.size() - 1];
//	auto& d_radius = radii[radii.size() - 1];
//
//	CCE(hipMalloc(reinterpret_cast<void**>(&d_center), sizeof(float3)));
//	CCE(hipMemcpy(d_center, &sphere.center, sizeof(float3), hipMemcpyHostToDevice));
//	CCE(hipMalloc(reinterpret_cast<void**>(&d_radius), sizeof(float)));
//	CCE(hipMemcpy(d_radius, &sphere.radius, sizeof(float), hipMemcpyHostToDevice));
//
//	sphere_input.sphereArray.vertexStrideInBytes = sizeof(float3);
//	sphere_input.sphereArray.numVertices = 1;
//	sphere_input.sphereArray.vertexBuffers = reinterpret_cast<hipDeviceptr_t*>(&d_center);
//
//	sphere_input.sphereArray.radiusStrideInBytes = sizeof(float);
//	sphere_input.sphereArray.radiusBuffers = reinterpret_cast<hipDeviceptr_t*>(&d_radius);
//
//	sphere_input.sphereArray.flags = flags;
//	sphere_input.sphereArray.numSbtRecords = 1;
//	sphere_input.sphereArray.sbtIndexOffsetBuffer = 0;
//	sphere_input.sphereArray.sbtIndexOffsetSizeInBytes = 0;
//	sphere_input.sphereArray.sbtIndexOffsetStrideInBytes = 0;
//
//	inputs.push_back(sphere_input);
//}
//
//static void prepare_cylinder_input(std::vector<OptixBuildInput>& inputs, Cylinder& cylinder, uint32_t* flags, std::vector<float*>& aabbs)
//{
//	OptixBuildInput cylinder_input{};
//    cylinder_input.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
//
//	aabbs.resize(aabbs.size() + 1);
//	auto& d_aabb = aabbs[aabbs.size() - 1];
//
//	Boundary boundary = cylinder.bound();
//	OptixAabb aabb{boundary.min_.x, boundary.min_.y, boundary.min_.z,
//			boundary.max_.x, boundary.max_.x, boundary.max_.x};
//
//	CCE(hipMalloc(reinterpret_cast<void**>(&d_aabb), sizeof(OptixAabb)));
//	CCE(hipMemcpy(d_aabb, &aabb, sizeof(OptixAabb), hipMemcpyHostToDevice));
//
//    cylinder_input.customPrimitiveArray.aabbBuffers = reinterpret_cast<hipDeviceptr_t*>(&d_aabb);
//    cylinder_input.customPrimitiveArray.numPrimitives = 1;
//
//    cylinder_input.customPrimitiveArray.flags = flags;
//    cylinder_input.customPrimitiveArray.numSbtRecords = 1;
//    cylinder_input.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
//    cylinder_input.customPrimitiveArray.sbtIndexOffsetSizeInBytes = 0;
//    cylinder_input.customPrimitiveArray.sbtIndexOffsetStrideInBytes = 0; 
//    cylinder_input.customPrimitiveArray.primitiveIndexOffset = 0;
//
//	inputs.push_back(cylinder_input);
//}
//
//static void prepare_triangle_input(std::vector<OptixBuildInput>& inputs, Model& model, uint32_t* flags)
//{
//	OptixBuildInput triangle_input{};
//	triangle_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
//
//	triangle_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
//	triangle_input.triangleArray.vertexStrideInBytes = sizeof(float3);
//	triangle_input.triangleArray.numVertices = static_cast<uint32_t>(model.vertex_count);
//	triangle_input.triangleArray.vertexBuffers = reinterpret_cast<hipDeviceptr_t*>(&model.d_vertices);
//
//	triangle_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
//	triangle_input.triangleArray.indexStrideInBytes = sizeof(int3);
//	triangle_input.triangleArray.numIndexTriplets = static_cast<uint32_t>(model.index_count);
//	triangle_input.triangleArray.indexBuffer = reinterpret_cast<hipDeviceptr_t>(model.d_indices);
//
//	triangle_input.triangleArray.flags = flags;
//	triangle_input.triangleArray.numSbtRecords = 1;
//	triangle_input.triangleArray.sbtIndexOffsetBuffer = 0;
//	triangle_input.triangleArray.sbtIndexOffsetSizeInBytes = 0;
//	triangle_input.triangleArray.sbtIndexOffsetStrideInBytes = 0;
//
//	inputs.push_back(triangle_input);
//}

OptixRenderer::OptixRenderer(const RenderInfo* render_info, WorldInfo* world_info, const SkyInfo* sky_info, const CameraInfo* camera_info)
	: render_info_(render_info), world_info_(world_info), sky_info_(sky_info), camera_info_(camera_info)
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
	auto blocks = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
	auto threads = dim3(thread_x, thread_y);

	init_optix();
	create_modules();
	create_programs();

	allocate_world();

	create_pipeline();
	create_sbt();

	CCE(hipMalloc(reinterpret_cast<void**>(&h_launch_params_.accumulation_buffer), sizeof(float4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_initial_), sizeof(uint4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&h_launch_params_.xoshiro_state), sizeof(uint4) * width * height));

	random_init<<<blocks, threads>>>(width, height, xoshiro_initial_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());

	CCE(hipMemcpy(h_launch_params_.xoshiro_state, xoshiro_initial_, sizeof(uint4) * width * height, hipMemcpyDeviceToDevice));

	if (sky_info_->h_hdr_data)
	{
		const uint64_t hdr_size = sizeof(float3) * sky_info_->hdr_width * sky_info_->hdr_height;
		CCE(hipMalloc(reinterpret_cast<void**>(&sky_info_->d_hdr_data), hdr_size));
		CCE(hipMemcpy(sky_info_->d_hdr_data, sky_info_->h_hdr_data, hdr_size, hipMemcpyHostToDevice));
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&d_launch_params_), sizeof(LaunchParams)));
	CCE(hipMemcpy(d_launch_params_, &h_launch_params_, sizeof(LaunchParams), hipMemcpyHostToDevice));
}

OptixRenderer::~OptixRenderer()
{
	if (sky_info_->h_hdr_data)
		CCE(hipFree(sky_info_->d_hdr_data));

	deallocate_world();

	CCE(hipFree(h_launch_params_.xoshiro_state));
	CCE(hipFree(xoshiro_initial_));
	CCE(hipFree(h_launch_params_.accumulation_buffer));

	CCE(hipFree(d_raygen_records_));
	CCE(hipFree(d_miss_records_));
	CCE(hipFree(d_hit_records_));

	CCE(hipFree(d_launch_params_));

	CCE(hipStreamDestroy(stream_));
	COE(optixDeviceContextDestroy(context_));

	hipDeviceReset();
}

void OptixRenderer::render()
{
	const auto frame_buffer = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	if (render_info_->progressive)
	{
		sbt_.raygenRecord = reinterpret_cast<hipDeviceptr_t>(d_raygen_records_);
		h_launch_params_.sampling_denominator = render_info_->frames_since_refresh;
	}
	else
	{
		sbt_.raygenRecord = reinterpret_cast<hipDeviceptr_t>(d_raygen_records_ + 1);
		CCE(hipMemset(frame_buffer, 0, render_info_->frame_size));
		h_launch_params_.sampling_denominator = render_info_->samples_per_pixel;
	}

	h_launch_params_.width = render_info_->width;
	h_launch_params_.height = render_info_->height;
	h_launch_params_.frame_buffer = frame_buffer;
	h_launch_params_.sky_info = *sky_info_;
	h_launch_params_.camera_info = *camera_info_;

	CCE(hipMemcpy(d_launch_params_, &h_launch_params_, sizeof(LaunchParams), hipMemcpyHostToDevice));

	COE(optixLaunch(
		pipeline_, stream_,
		reinterpret_cast<hipDeviceptr_t>(d_launch_params_),
		sizeof(LaunchParams),
		&sbt_,
		h_launch_params_.width,
		h_launch_params_.height,
		1));

	CCE(hipDeviceSynchronize());
	CCE(hipGetLastError());

	CCE(hipFree(frame_buffer));
}

void OptixRenderer::refresh_buffer()
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;

	CCE(hipMemset(h_launch_params_.accumulation_buffer, 0, sizeof(float4) * width * height));
	CCE(hipMemcpy(h_launch_params_.xoshiro_state, xoshiro_initial_, sizeof(uint4) * width * height, hipMemcpyDeviceToDevice));
}

void OptixRenderer::refresh_object(int32_t index) const
{
}

void OptixRenderer::refresh_material(int32_t index) const
{
}

void OptixRenderer::refresh_texture(int32_t index) const
{
}

void OptixRenderer::recreate_image()
{
	const uint32_t width = render_info_->width;
	const uint32_t height = render_info_->height;
	constexpr int32_t thread_x = 16;
	constexpr int32_t thread_y = 16;
	auto blocks = dim3((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
	auto threads = dim3(thread_x, thread_y);

	CCE(hipFree(h_launch_params_.xoshiro_state));
	CCE(hipFree(xoshiro_initial_));
	CCE(hipFree(h_launch_params_.accumulation_buffer));
	CCE(hipMalloc(reinterpret_cast<void**>(&h_launch_params_.accumulation_buffer), sizeof(float4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&xoshiro_initial_), sizeof(uint4) * width * height));
	CCE(hipMalloc(reinterpret_cast<void**>(&h_launch_params_.xoshiro_state), sizeof(uint4) * width * height));

	random_init<<<blocks, threads>>>(width, height, xoshiro_initial_);
	CCE(hipGetLastError());
	CCE(hipDeviceSynchronize());
}

void OptixRenderer::recreate_sky()
{
	CCE(hipFree(sky_info_->d_hdr_data));

	if (sky_info_->h_hdr_data)
	{
		const uint64_t hdr_size = sizeof(float3) * sky_info_->hdr_width * sky_info_->hdr_height;
		CCE(hipMalloc(reinterpret_cast<void**>(&sky_info_->d_hdr_data), hdr_size));
		CCE(hipMemcpy(sky_info_->d_hdr_data, sky_info_->h_hdr_data, hdr_size, hipMemcpyHostToDevice));
	}
	else
		sky_info_->d_hdr_data = nullptr;
}

void OptixRenderer::map_frame_memory()
{
	const auto frame_buffer = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	CCE(hipMemcpy(render_info_->frame_data, frame_buffer, render_info_->frame_size, hipMemcpyDeviceToHost));
	CCE(hipFree(frame_buffer));
}

void OptixRenderer::allocate_world()
{
	for (auto& texture : world_info_->textures_)
	{
		if (texture.type == TextureType::IMAGE)
		{
			const auto image_data = &texture.image;
			const uint64_t image_size = sizeof(float) * image_data->width * image_data->height * 3;

			CCE(hipMalloc(reinterpret_cast<void**>(&image_data->d_data), image_size));
			CCE(hipMemcpy(image_data->d_data, image_data->h_data, image_size, hipMemcpyHostToDevice));
		}
	}

	for (auto& object : world_info_->objects_)
	{
		if (object.type == ObjectType::MODEL)
		{
			const auto model_data = &object.model;

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_vertices), model_data->vertex_count * sizeof(float3)));
			CCE(hipMemcpy(model_data->d_vertices, model_data->h_vertices, model_data->vertex_count * sizeof(float3), hipMemcpyHostToDevice));

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_indices), model_data->index_count * sizeof(uint3)));
			CCE(hipMemcpy(model_data->d_indices, model_data->h_indices, model_data->index_count * sizeof(uint3), hipMemcpyHostToDevice));

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_normals), model_data->vertex_count * sizeof(float3)));
			CCE(hipMemcpy(model_data->d_normals, model_data->h_normals, model_data->vertex_count * sizeof(float3), hipMemcpyHostToDevice));

			CCE(hipMalloc(reinterpret_cast<void**>(&model_data->d_uv), model_data->vertex_count * sizeof(float2)));
			CCE(hipMemcpy(model_data->d_uv, model_data->h_uv, model_data->vertex_count * sizeof(float2), hipMemcpyHostToDevice));
		}
	}

	std::vector<OptixBuildInput> sphere_inputs{}, cylinder_inputs{}, triangle_inputs{};
	std::vector<float3*> centers_buffer{};
	std::vector<float*> radii_buffer{}, aabbs_buffer{};
	uint32_t input_flags[1] = { OPTIX_BUILD_FLAG_NONE };

	/*for (auto& object : world_info_->objects_)
	{
		if (object.type == ObjectType::SPHERE)
			prepare_sphere_input(sphere_inputs, object.sphere, input_flags, centers_buffer, radii_buffer);
		else if (object.type == ObjectType::CYLINDER)
			prepare_cylinder_input(cylinder_inputs, object.cylinder, input_flags, aabbs_buffer);
		else if (object.type == ObjectType::MODEL)
			prepare_triangle_input(triangle_inputs, object.model, input_flags);
	}*/

	build_gases(sphere_inputs, cylinder_inputs, triangle_inputs, centers_buffer, radii_buffer, aabbs_buffer, input_flags);

	for (const auto& buffer : centers_buffer)
		CCE(hipFree(buffer));

	for (const auto& buffer : radii_buffer)
		CCE(hipFree(buffer));

	for (const auto& buffer : aabbs_buffer)
		CCE(hipFree(buffer));
}

void OptixRenderer::deallocate_world() const
{
	CCE(hipFree(triangle_gas_buffer_));
	//CCE(hipFree(sphere_gas_buffer_));
	//CCE(hipFree(cylinder_gas_buffer_));

	for (const auto& object : world_info_->objects_)
	{
		if (object.type == ObjectType::MODEL)
		{
			CCE(hipFree(object.model.d_vertices));
			CCE(hipFree(object.model.d_indices));
			CCE(hipFree(object.model.d_normals));
			CCE(hipFree(object.model.d_uv));
		}
	}

	for (const auto& texture : world_info_->textures_)
	{
		if (texture.type == TextureType::IMAGE)
			CCE(hipFree(texture.image.d_data));
	}
}

void OptixRenderer::init_optix()
{
	COE(optixInit());

	OptixDeviceContextOptions options{};

#ifdef _DEBUG
	options.validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL;
	options.logCallbackFunction = &context_log;
	options.logCallbackLevel = 4;
#endif

	const hipCtx_t cuda_context = nullptr;

	CCE(hipStreamCreate(&stream_));
	COE(optixDeviceContextCreate(cuda_context, &options, &context_));
}

void OptixRenderer::create_modules()
{
	module_compile_options_.maxRegisterCount = 50;
#ifdef _DEBUG
	module_compile_options_.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
	module_compile_options_.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
#else
	module_compile_options_.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
	module_compile_options_.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;
#endif

	pipeline_compile_options_.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
	pipeline_compile_options_.usesPrimitiveTypeFlags = OPTIX_PRIMITIVE_TYPE_FLAGS_CUSTOM |
		OPTIX_PRIMITIVE_TYPE_FLAGS_SPHERE;
	pipeline_compile_options_.usesMotionBlur = false;
	pipeline_compile_options_.numPayloadValues = 2;
	pipeline_compile_options_.numAttributeValues = 2;
#ifdef _DEBUG
	pipeline_compile_options_.exceptionFlags = OPTIX_EXCEPTION_FLAG_DEBUG;
#else
	pipeline_compile_options_.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
#endif
	pipeline_compile_options_.pipelineLaunchParamsVariableName = "launch_params";

	const std::string shader = read_shader("OptixPrograms.optixir");

	COE(optixModuleCreate(
		context_,
		&module_compile_options_,
		&pipeline_compile_options_,
		shader.c_str(),
		shader.size(),
		nullptr, nullptr,
		&module_));
}

void OptixRenderer::create_programs()
{
	raygen_programs_.resize(2);
	OptixProgramGroupOptions rg_options = {};
	OptixProgramGroupDesc p_rg_desc = {};
	p_rg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
	p_rg_desc.raygen.module = module_;
	p_rg_desc.raygen.entryFunctionName = "__raygen__render_progressive";

	COE(optixProgramGroupCreate(
		context_,
		&p_rg_desc,
		1,
		&rg_options,
		nullptr, nullptr,
		raygen_programs_.data()));

	OptixProgramGroupDesc s_rg_desc = {};
	s_rg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
	s_rg_desc.raygen.module = module_;
	s_rg_desc.raygen.entryFunctionName = "__raygen__render_static";

	COE(optixProgramGroupCreate(
		context_,
		&s_rg_desc,
		1,
		&rg_options,
		nullptr, nullptr,
		raygen_programs_.data() + 1));

	miss_programs_.resize(1);
	OptixProgramGroupOptions m_options = {};
	OptixProgramGroupDesc m_desc = {};
	m_desc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
	m_desc.miss.module = module_;
	m_desc.miss.entryFunctionName = "__miss__radiance";

	COE(optixProgramGroupCreate(
		context_,
		&m_desc,
		1,
		&m_options,
		nullptr, nullptr,
		miss_programs_.data()));

	hit_programs_.resize(3);
	OptixProgramGroupOptions hg_options = {};
	OptixModule sphere_is_module{};
	OptixBuiltinISOptions sphere_is_options = {};
	sphere_is_options.builtinISModuleType = OPTIX_PRIMITIVE_TYPE_SPHERE;
	sphere_is_options.usesMotionBlur = 0;

	COE(optixBuiltinISModuleGet(
		context_, 
		&module_compile_options_, 
		&pipeline_compile_options_, 
		&sphere_is_options, 
		&sphere_is_module));

	OptixProgramGroupDesc s_hg_desc = {};
	s_hg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
	s_hg_desc.hitgroup.moduleCH = module_;
	s_hg_desc.hitgroup.entryFunctionNameCH = "__closesthit__sphere";
	s_hg_desc.hitgroup.moduleIS = sphere_is_module;

	COE(optixProgramGroupCreate(
		context_,
		&s_hg_desc,
		1,
		&hg_options,
		nullptr, nullptr,
		hit_programs_.data()));

	OptixProgramGroupDesc c_hg_desc = {};
	c_hg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
	c_hg_desc.hitgroup.moduleCH = module_;
	c_hg_desc.hitgroup.entryFunctionNameCH = "__closesthit__cylinder";
	c_hg_desc.hitgroup.moduleIS = module_;
	c_hg_desc.hitgroup.entryFunctionNameIS = "__intersection__cylinder";

	COE(optixProgramGroupCreate(
		context_,
		&c_hg_desc,
		1,
		&hg_options,
		nullptr, nullptr,
		hit_programs_.data() + 1));

	OptixProgramGroupDesc t_hg_desc = {};
	t_hg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
	t_hg_desc.hitgroup.moduleCH = module_;
	t_hg_desc.hitgroup.entryFunctionNameCH = "__closesthit__triangle";

	COE(optixProgramGroupCreate(
		context_,
		&t_hg_desc,
		1,
		&hg_options,
		nullptr, nullptr,
		hit_programs_.data() + 2));
}

void OptixRenderer::create_pipeline()
{
	std::vector<OptixProgramGroup> program_groups;

	program_groups.reserve(program_groups.size() + raygen_programs_.size());
	for (auto pg : raygen_programs_)
		program_groups.push_back(pg);

	program_groups.reserve(program_groups.size() + miss_programs_.size());
	for (auto pg : miss_programs_)
		program_groups.push_back(pg);

	program_groups.reserve(program_groups.size() + hit_programs_.size());
	for (auto pg : hit_programs_)
		program_groups.push_back(pg);

	const OptixPipelineLinkOptions pipeline_link_options{ static_cast<uint32_t>(render_info_->max_depth) };

	COE(optixPipelineCreate(
		context_,
		&pipeline_compile_options_,
		&pipeline_link_options,
		program_groups.data(),
		static_cast<uint32_t>(program_groups.size()),
		nullptr, nullptr,
		&pipeline_));

	COE(optixPipelineSetStackSize(pipeline_, 2 * 1024, 2 * 1024, 2 * 1024, 1));
}

void OptixRenderer::build_gases(std::vector<OptixBuildInput>& sphere_inputs, std::vector<OptixBuildInput>& cylinder_inputs, 
	std::vector<OptixBuildInput>& triangle_inputs, std::vector<float3*>& centers, 
	std::vector<float*>& radii, std::vector<float*>& aabbs, uint32_t* flags)
{
	for (auto& object : world_info_->objects_)
	{
		if (object.type == ObjectType::SPHERE)
		{
			OptixBuildInput sphere_input{};
			sphere_input.type = OPTIX_BUILD_INPUT_TYPE_SPHERES;

			centers.resize(centers.size() + 1);
			radii.resize(radii.size() + 1);
			auto& d_center = centers[centers.size() - 1];
			auto& d_radius = radii[radii.size() - 1];

			CCE(hipMalloc(reinterpret_cast<void**>(&d_center), sizeof(float3)));
			CCE(hipMemcpy(d_center, &object.sphere.center, sizeof(float3), hipMemcpyHostToDevice));
			CCE(hipMalloc(reinterpret_cast<void**>(&d_radius), sizeof(float)));
			CCE(hipMemcpy(d_radius, &object.sphere.radius, sizeof(float), hipMemcpyHostToDevice));

			sphere_input.sphereArray.vertexStrideInBytes = sizeof(float3);
			sphere_input.sphereArray.numVertices = 1;
			sphere_input.sphereArray.vertexBuffers = reinterpret_cast<hipDeviceptr_t*>(centers.data());

			sphere_input.sphereArray.radiusStrideInBytes = sizeof(float);
			sphere_input.sphereArray.radiusBuffers = reinterpret_cast<hipDeviceptr_t*>(radii.data());

			sphere_input.sphereArray.flags = flags;
			sphere_input.sphereArray.numSbtRecords = 1;
			sphere_input.sphereArray.sbtIndexOffsetBuffer = 0;
			sphere_input.sphereArray.sbtIndexOffsetSizeInBytes = 0;
			sphere_input.sphereArray.sbtIndexOffsetStrideInBytes = 0;

			sphere_inputs.push_back(sphere_input);
		}
		else if (object.type == ObjectType::CYLINDER)
		{
			OptixBuildInput cylinder_input{};
			cylinder_input.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;

			aabbs.resize(aabbs.size() + 1);
			auto& d_aabb = aabbs[aabbs.size() - 1];

			Boundary boundary = object.cylinder.bound();
			OptixAabb aabb{boundary.min_.x, boundary.min_.y, boundary.min_.z,
					boundary.max_.x, boundary.max_.x, boundary.max_.x};

			CCE(hipMalloc(reinterpret_cast<void**>(&d_aabb), sizeof(OptixAabb)));
			CCE(hipMemcpy(d_aabb, &aabb, sizeof(OptixAabb), hipMemcpyHostToDevice));

			cylinder_input.customPrimitiveArray.aabbBuffers = reinterpret_cast<hipDeviceptr_t*>(aabbs.data());
			cylinder_input.customPrimitiveArray.numPrimitives = 1;

			cylinder_input.customPrimitiveArray.flags = flags;
			cylinder_input.customPrimitiveArray.numSbtRecords = 1;
			cylinder_input.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
			cylinder_input.customPrimitiveArray.sbtIndexOffsetSizeInBytes = 0;
			cylinder_input.customPrimitiveArray.sbtIndexOffsetStrideInBytes = 0; 
			cylinder_input.customPrimitiveArray.primitiveIndexOffset = 0;

			cylinder_inputs.push_back(cylinder_input);
		}
		else if (object.type == ObjectType::MODEL)
		{
			OptixBuildInput triangle_input{};
			triangle_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

			triangle_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
			triangle_input.triangleArray.vertexStrideInBytes = sizeof(float3);
			triangle_input.triangleArray.numVertices = static_cast<uint32_t>(object.model.vertex_count);
			triangle_input.triangleArray.vertexBuffers = reinterpret_cast<hipDeviceptr_t*>(&object.model.d_vertices);

			triangle_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
			triangle_input.triangleArray.indexStrideInBytes = sizeof(int3);
			triangle_input.triangleArray.numIndexTriplets = static_cast<uint32_t>(object.model.index_count);
			triangle_input.triangleArray.indexBuffer = reinterpret_cast<hipDeviceptr_t>(object.model.d_indices);

			triangle_input.triangleArray.flags = flags;
			triangle_input.triangleArray.numSbtRecords = 1;
			triangle_input.triangleArray.sbtIndexOffsetBuffer = 0;
			triangle_input.triangleArray.sbtIndexOffsetSizeInBytes = 0;
			triangle_input.triangleArray.sbtIndexOffsetStrideInBytes = 0;

			triangle_inputs.push_back(triangle_input);
		}
	}

	//h_launch_params_.traversable = build_gas(sphere_inputs, sphere_gas_buffer_);
	//h_launch_params_.traversable = build_gas(cylinder_inputs, cylinder_gas_buffer_);
	h_launch_params_.traversable = build_gas(triangle_inputs, triangle_gas_buffer_);
}

OptixTraversableHandle OptixRenderer::build_gas(const std::vector<OptixBuildInput>& build_inputs, void*& gas_buffer) const
{
	OptixTraversableHandle gas_handle = {0};

	OptixAccelBuildOptions accel_options = {};
	accel_options.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	accel_options.motionOptions.numKeys = 1;
	accel_options.operation = OPTIX_BUILD_OPERATION_BUILD;

	OptixAccelBufferSizes blas_buffer_sizes;
	COE(optixAccelComputeMemoryUsage(
		context_,
		&accel_options,
		build_inputs.data(),
		static_cast<uint32_t>(build_inputs.size()),
		&blas_buffer_sizes));

	uint64_t* compacted_size_buffer = nullptr;
	CCE(hipMalloc(reinterpret_cast<void**>(&compacted_size_buffer), sizeof(uint64_t)));

	OptixAccelEmitDesc emit_desc;
	emit_desc.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
	emit_desc.result = reinterpret_cast<hipDeviceptr_t>(compacted_size_buffer);

	void* temp_buffer = nullptr;
	CCE(hipMalloc(&temp_buffer, blas_buffer_sizes.tempSizeInBytes));

	void* output_buffer;
	CCE(hipMalloc(&output_buffer, blas_buffer_sizes.outputSizeInBytes));

	COE(optixAccelBuild(
		context_,
		nullptr,
		&accel_options,
		build_inputs.data(),
		static_cast<uint32_t>(build_inputs.size()),
		reinterpret_cast<hipDeviceptr_t>(temp_buffer),
		blas_buffer_sizes.tempSizeInBytes,
		reinterpret_cast<hipDeviceptr_t>(output_buffer),
		blas_buffer_sizes.outputSizeInBytes,
		&gas_handle,
		&emit_desc, 1));

	CCE(hipDeviceSynchronize());
	CCE(hipGetLastError());

	uint64_t compacted_size;
	CCE(hipMemcpy(&compacted_size, compacted_size_buffer, sizeof(uint64_t), hipMemcpyDeviceToHost));

	CCE(hipMalloc(&gas_buffer, compacted_size));
	COE(optixAccelCompact(
		context_,
		nullptr,
		gas_handle,
		reinterpret_cast<hipDeviceptr_t>(gas_buffer),
		compacted_size,
		&gas_handle));

	CCE(hipDeviceSynchronize());
	CCE(hipGetLastError());

	CCE(hipFree(output_buffer));
	CCE(hipFree(temp_buffer));
	CCE(hipFree(compacted_size_buffer));

	return gas_handle;
}

OptixTraversableHandle OptixRenderer::build_ias()
{
	OptixTraversableHandle as_handle{};

	return as_handle;
}

void OptixRenderer::create_sbt()
{
	std::vector<SbtRecord<RayGenData>> raygen_records;
	for (const auto& raygen_program : raygen_programs_)
	{
		SbtRecord<RayGenData> rec{};
		COE(optixSbtRecordPackHeader(raygen_program, &rec));
		raygen_records.push_back(rec);
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&d_raygen_records_), raygen_records.size() * sizeof(SbtRecord<RayGenData>)));
	CCE(hipMemcpy(d_raygen_records_, raygen_records.data(), raygen_records.size() * sizeof(SbtRecord<RayGenData>), hipMemcpyHostToDevice));

	std::vector<SbtRecord<MissData>> miss_records;
	for (const auto& miss_program : miss_programs_)
	{
		SbtRecord<MissData> rec{};
		COE(optixSbtRecordPackHeader(miss_program, &rec));
		miss_records.push_back(rec);
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&d_miss_records_), miss_records.size() * sizeof(SbtRecord<MissData>)));
	CCE(hipMemcpy(d_miss_records_, miss_records.data(), miss_records.size() * sizeof(SbtRecord<MissData>), hipMemcpyHostToDevice));

	sbt_.missRecordBase = reinterpret_cast<hipDeviceptr_t>(d_miss_records_);
	sbt_.missRecordStrideInBytes = sizeof(SbtRecord<MissData>);
	sbt_.missRecordCount = static_cast<uint32_t>(miss_records.size());

	std::vector<SbtRecord<HitGroupData>> hitgroup_records;
	for (const auto& object : world_info_->objects_)
	{
		SbtRecord<HitGroupData> rec{};
		COE(optixSbtRecordPackHeader(hit_programs_[enum_cast(object.type) - 1], &rec));
		rec.data.texture = world_info_->textures_[object.texture_id];
		rec.data.material = world_info_->materials_[object.material_id];
		rec.data.object = object;
		hitgroup_records.push_back(rec);
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&d_hit_records_), hitgroup_records.size() * sizeof(SbtRecord<HitGroupData>)));
	CCE(hipMemcpy(d_hit_records_, hitgroup_records.data(), hitgroup_records.size() * sizeof(SbtRecord<HitGroupData>), hipMemcpyHostToDevice));

	sbt_.hitgroupRecordBase = reinterpret_cast<hipDeviceptr_t>(d_hit_records_);
	sbt_.hitgroupRecordStrideInBytes = sizeof(SbtRecord<HitGroupData>);
	sbt_.hitgroupRecordCount = static_cast<uint32_t>(hitgroup_records.size());
}
