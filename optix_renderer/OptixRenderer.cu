#include "hip/hip_runtime.h"
// ReSharper disable once CppPrecompiledHeaderIsNotIncluded
#include "OptixRenderer.cuh"

#include "../common/Utils.hpp"

#include <optix_stubs.h>
#include <optix_function_table_definition.h>

#ifdef _DEBUG
static void context_log(unsigned int level, const char* tag, const char* message, void*)
{
	printf("[%u][%s]: %s\n", level, tag, message);
}
#endif

OptixRenderer::OptixRenderer(const RenderInfo* render_info, const WorldInfo* world_info, const SkyInfo* sky_info)
	: render_info_(render_info), world_info_(world_info), sky_info_(sky_info)
{
	init_optix();
	create_modules();
	create_programs();
	create_pipeline();
	create_sbt();
	host_launch_params_.traversable = build_as();

	CCE(hipMalloc(reinterpret_cast<void**>(&device_launch_params_), sizeof(LaunchParams)));
	CCE(hipMemcpy(device_launch_params_, &host_launch_params_, sizeof(LaunchParams), hipMemcpyHostToDevice));
}

OptixRenderer::~OptixRenderer()
{
	CCE(hipStreamDestroy(stream_));
	CCE(hipFree(device_launch_params_));
	CCE(hipFree(device_raygen_records_));
	CCE(hipFree(device_miss_records_));
	CCE(hipFree(device_hit_records_));
	CCE(hipFree(device_index_buffer_));
	CCE(hipFree(device_vertex_buffer_));
	CCE(hipFree(device_as_buffer_));

	hipDeviceReset();
}

void OptixRenderer::render_static()
{
}

void OptixRenderer::render_progressive()
{
	const auto frame_buffer = static_cast<float4*>(fetch_external_memory(render_info_->frame_handle, render_info_->frame_size));

	host_launch_params_.width = render_info_->width;
	host_launch_params_.height = render_info_->height;
	host_launch_params_.frame_buffer = frame_buffer;

	CCE(hipMemcpy(device_launch_params_, &host_launch_params_, sizeof(LaunchParams), hipMemcpyHostToDevice));

	COE(optixLaunch(
		pipeline_, stream_,
		reinterpret_cast<hipDeviceptr_t>(device_launch_params_),
		sizeof(LaunchParams),
		&sbt_,
		host_launch_params_.width,
		host_launch_params_.height,
		1));

	CCE(hipDeviceSynchronize());
	CCE(hipGetLastError());

	CCE(hipFree(frame_buffer));
}

void OptixRenderer::refresh_buffer()
{
}

void OptixRenderer::refresh_camera()
{
}

void OptixRenderer::refresh_object(int32_t index) const
{
}

void OptixRenderer::refresh_material(int32_t index) const
{
}

void OptixRenderer::refresh_texture(int32_t index) const
{
}

void OptixRenderer::recreate_camera()
{
}

void OptixRenderer::recreate_image()
{
}

void OptixRenderer::recreate_sky()
{
}

void OptixRenderer::allocate_world()
{
}

void OptixRenderer::deallocate_world() const
{
}

void OptixRenderer::init_optix()
{
	COE(optixInit());

	OptixDeviceContextOptions options{};

#ifdef _DEBUG
	options.logCallbackFunction = &context_log;
	options.logCallbackLevel = 4;
#endif

	const hipCtx_t cuda_context = nullptr;

	CCE(hipStreamCreate(&stream_));
	COE(optixDeviceContextCreate(cuda_context, &options, &context_));
}

void OptixRenderer::create_modules()
{
	OptixModuleCompileOptions module_compile_options{};
	module_compile_options.maxRegisterCount = 50;
#ifdef _DEBUG
	module_compile_options.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
	module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
#else
	module_compile_options.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
	module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;
#endif

	OptixPipelineCompileOptions pipeline_compile_options{};
	pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
	pipeline_compile_options.usesMotionBlur = false;
	pipeline_compile_options.numPayloadValues = 2;
	pipeline_compile_options.numAttributeValues = 2;
	pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
	pipeline_compile_options.pipelineLaunchParamsVariableName = "launch_params";

	const std::string shader = read_shader("OptixPrograms.optixir");

	COE(optixModuleCreate(
		context_,
		&module_compile_options,
		&pipeline_compile_options,
		shader.c_str(),
		shader.size(),
		nullptr, nullptr,
		&module_));
}

void OptixRenderer::create_programs()
{
	raygen_programs_.resize(1);
	OptixProgramGroupOptions rg_options = {};
	OptixProgramGroupDesc rg_desc = {};
	rg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
	rg_desc.raygen.module = module_;
	rg_desc.raygen.entryFunctionName = "__raygen__render";

	COE(optixProgramGroupCreate(
		context_,
		&rg_desc,
		1,
		&rg_options,
		nullptr, nullptr,
		raygen_programs_.data()));

	miss_programs_.resize(1);
	OptixProgramGroupOptions m_options = {};
	OptixProgramGroupDesc m_desc = {};
	m_desc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
	m_desc.miss.module = module_;
	m_desc.miss.entryFunctionName = "__miss__radiance";

	COE(optixProgramGroupCreate(
		context_,
		&m_desc,
		1,
		&m_options,
		nullptr, nullptr,
		miss_programs_.data()));

	hit_programs_.resize(1);
	OptixProgramGroupOptions hg_options = {};
	OptixProgramGroupDesc hg_desc = {};
	hg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
	hg_desc.hitgroup.moduleCH = module_;
	hg_desc.hitgroup.entryFunctionNameCH = "__closesthit__radiance";
	hg_desc.hitgroup.moduleAH = module_;
	hg_desc.hitgroup.entryFunctionNameAH = "__anyhit__radiance";

	COE(optixProgramGroupCreate(
		context_,
		&hg_desc,
		1,
		&hg_options,
		nullptr, nullptr,
		hit_programs_.data()));
}

void OptixRenderer::create_pipeline()
{
	std::vector<OptixProgramGroup> program_groups;

	program_groups.reserve(program_groups.size() + raygen_programs_.size());
	for (auto pg : raygen_programs_)
		program_groups.push_back(pg);

	program_groups.reserve(program_groups.size() + miss_programs_.size());
	for (auto pg : miss_programs_)
		program_groups.push_back(pg);

	program_groups.reserve(program_groups.size() + hit_programs_.size());
	for (auto pg : hit_programs_)
		program_groups.push_back(pg);

	OptixPipelineCompileOptions pipeline_compile_options{};
	pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
	pipeline_compile_options.usesMotionBlur = false;
	pipeline_compile_options.numPayloadValues = 2;
	pipeline_compile_options.numAttributeValues = 2;
	pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
	pipeline_compile_options.pipelineLaunchParamsVariableName = "launch_params";

	constexpr OptixPipelineLinkOptions pipeline_link_options{ 2 };

	COE(optixPipelineCreate(
		context_,
		&pipeline_compile_options,
		&pipeline_link_options,
		program_groups.data(),
		static_cast<uint32_t>(program_groups.size()),
		nullptr, nullptr,
		&pipeline_));

	COE(optixPipelineSetStackSize(pipeline_, 2 * 1024, 2 * 1024, 2 * 1024, 1));
}

void OptixRenderer::create_sbt()
{
	std::vector<SbtRecord<RayGenData>> raygen_records;
	for (const auto& raygen_program : raygen_programs_)
	{
		SbtRecord<RayGenData> rec{};
		COE(optixSbtRecordPackHeader(raygen_program, &rec));
		raygen_records.push_back(rec);
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&device_raygen_records_), raygen_records.size() * sizeof(SbtRecord<RayGenData>)));
	CCE(hipMemcpy(device_raygen_records_, raygen_records.data(), raygen_records.size() * sizeof(SbtRecord<RayGenData>), hipMemcpyHostToDevice));

	sbt_.raygenRecord = reinterpret_cast<hipDeviceptr_t>(device_raygen_records_);

	std::vector<SbtRecord<MissData>> miss_records;
	for (const auto& miss_program : miss_programs_)
	{
		SbtRecord<MissData> rec{};
		COE(optixSbtRecordPackHeader(miss_program, &rec));
		miss_records.push_back(rec);
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&device_miss_records_), miss_records.size() * sizeof(SbtRecord<MissData>)));
	CCE(hipMemcpy(device_miss_records_, miss_records.data(), miss_records.size() * sizeof(SbtRecord<MissData>), hipMemcpyHostToDevice));

	sbt_.missRecordBase = reinterpret_cast<hipDeviceptr_t>(device_miss_records_);
	sbt_.missRecordStrideInBytes = sizeof(SbtRecord<MissData>);
	sbt_.missRecordCount = static_cast<uint32_t>(miss_records.size());

	constexpr int32_t num_objects = 1;
	std::vector<SbtRecord<HitGroupData>> hitgroup_records;
	for (int i = 0; i < num_objects; i++)
	{
		constexpr int32_t object_type = 0;
		SbtRecord<HitGroupData> rec{};
		COE(optixSbtRecordPackHeader(hit_programs_[object_type], &rec));
		rec.data.object_id = i;
		hitgroup_records.push_back(rec);
	}

	CCE(hipMalloc(reinterpret_cast<void**>(&device_hit_records_), hitgroup_records.size() * sizeof(SbtRecord<HitGroupData>)));
	CCE(hipMemcpy(device_hit_records_, hitgroup_records.data(), hitgroup_records.size() * sizeof(SbtRecord<HitGroupData>), hipMemcpyHostToDevice));

	sbt_.hitgroupRecordBase = reinterpret_cast<hipDeviceptr_t>(device_hit_records_);
	sbt_.hitgroupRecordStrideInBytes = sizeof(SbtRecord<HitGroupData>);
	sbt_.hitgroupRecordCount = static_cast<uint32_t>(hitgroup_records.size());
}

OptixTraversableHandle OptixRenderer::build_as()
{
	std::vector<float3> vertices;
	std::vector<int3> indices;
	vertices.push_back(make_float3(-1.0f, 0.0f, 0.0f));
	vertices.push_back(make_float3(1.0f, 0.0f, 0.0f));
	vertices.push_back(make_float3(0.0f, 2.0f, 0.0f));
	indices.push_back(make_int3(0, 1, 2));

	CCE(hipMalloc(reinterpret_cast<void**>(&device_vertex_buffer_), vertices.size() * sizeof(float3)));
	CCE(hipMemcpy(device_vertex_buffer_, vertices.data(), vertices.size() * sizeof(float3), hipMemcpyHostToDevice));
	CCE(hipMalloc(reinterpret_cast<void**>(&device_index_buffer_), indices.size() * sizeof(uint3)));
	CCE(hipMemcpy(device_index_buffer_, indices.data(), indices.size() * sizeof(uint3), hipMemcpyHostToDevice));

	OptixTraversableHandle as_handle{ 0 };

	OptixBuildInput triangle_input = {};
	triangle_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

	triangle_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
	triangle_input.triangleArray.vertexStrideInBytes = sizeof(float3);
	triangle_input.triangleArray.numVertices = static_cast<uint32_t>(vertices.size());
	triangle_input.triangleArray.vertexBuffers = reinterpret_cast<hipDeviceptr_t*>(&device_vertex_buffer_);

	triangle_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
	triangle_input.triangleArray.indexStrideInBytes = sizeof(int3);
	triangle_input.triangleArray.numIndexTriplets = static_cast<uint32_t>(indices.size());
	triangle_input.triangleArray.indexBuffer = reinterpret_cast<hipDeviceptr_t>(device_index_buffer_);

	uint32_t triangle_input_flags[1] = { 0 };

	triangle_input.triangleArray.flags = triangle_input_flags;
	triangle_input.triangleArray.numSbtRecords = 1;
	triangle_input.triangleArray.sbtIndexOffsetBuffer = 0;
	triangle_input.triangleArray.sbtIndexOffsetSizeInBytes = 0;
	triangle_input.triangleArray.sbtIndexOffsetStrideInBytes = 0;

	OptixAccelBuildOptions accel_options = {};
	accel_options.buildFlags = OPTIX_BUILD_FLAG_NONE | OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	accel_options.motionOptions.numKeys = 1;
	accel_options.operation = OPTIX_BUILD_OPERATION_BUILD;

	OptixAccelBufferSizes blas_buffer_sizes;
	COE(optixAccelComputeMemoryUsage(
		context_,
		&accel_options,
		&triangle_input,
		1,
		&blas_buffer_sizes));

	uint64_t* compacted_size_buffer = nullptr;
	CCE(hipMalloc(reinterpret_cast<void**>(&compacted_size_buffer), sizeof(uint64_t)));

	OptixAccelEmitDesc emit_desc;
	emit_desc.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
	emit_desc.result = reinterpret_cast<hipDeviceptr_t>(compacted_size_buffer);

	void* temp_buffer = nullptr;
	CCE(hipMalloc(&temp_buffer, blas_buffer_sizes.tempSizeInBytes));

	void* output_buffer;
	CCE(hipMalloc(&output_buffer, blas_buffer_sizes.outputSizeInBytes));

	COE(optixAccelBuild(context_,
		nullptr,
		&accel_options,
		&triangle_input,
		1,
		reinterpret_cast<hipDeviceptr_t>(temp_buffer),
		blas_buffer_sizes.tempSizeInBytes,
		reinterpret_cast<hipDeviceptr_t>(output_buffer),
		blas_buffer_sizes.outputSizeInBytes,
		&as_handle,
		&emit_desc, 1));

	CCE(hipDeviceSynchronize());
	CCE(hipGetLastError());

	uint64_t compacted_size;
	CCE(hipMemcpy(&compacted_size, compacted_size_buffer, sizeof(uint64_t), hipMemcpyDeviceToHost));

	CCE(hipMalloc(&device_as_buffer_, compacted_size));
	COE(optixAccelCompact(
		context_,
		nullptr,
		as_handle,
		reinterpret_cast<hipDeviceptr_t>(device_as_buffer_),
		compacted_size,
		&as_handle));

	CCE(hipDeviceSynchronize());
	CCE(hipGetLastError());

	CCE(hipFree(output_buffer));
	CCE(hipFree(temp_buffer));
	CCE(hipFree(compacted_size_buffer));

	return as_handle;
}
